#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */



#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

#include "rasterize.h"


#include <util/tiny_gltf_loader.h>

namespace {

	struct VertexOut {
		glm::vec4 pos;

		glm::vec3 eyePos;	// for shading
		glm::vec3 eyeNor;	// normal will go wrong after perspective transform

		//glm::vec3 col;
		glm::vec2 texcoord0;
	};

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};
	struct Fragment {
		glm::vec3 color;
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Vertex Out, changing for each frame
		VertexOut* dev_verticesOut;

		//TODO: add more attributes when necessary
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;


/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}



// TODO: delete me for assignment !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
static int * dev_depth = NULL;
__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);

		depth[index] = INT_MAX;

	}


}
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!1





/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");

	// TODO delete
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height *sizeof(int));
}











// Buffer State

// Attribute State (bufferview pointer, byte offset, byte stride, count(vec2/vec3), primitive type)

// 1. for mesh, for each primitive, create device buffer for indices and attributes (accessor), and bind all attribute(acessor) state
// 2. (kern) vertex shader (transform position)
// 3. for each primitive, do primitive assembly ( each attribute buffer => Primitive * dev_primitives)






/**
* kern function with support for stride to sometimes replace hipMemcpy
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		for (int j = 0; j < componentTypeByteSize; j++) {
			dev_dst[i * componentTypeByteSize + j] = dev_src[byteOffset + i * (byteStride == 0 ? componentTypeByteSize : byteStride) + j];
		}
	}
	

}


void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			//const tinygltf::Buffer &buffer = scene.buffers[bufferView.buffer];
			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			// ? __constant__
			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each meshes: for each primitive: build device buffer of indices, materail, and each attributes
	{
		std::map<std::string, tinygltf::Mesh>::const_iterator it(scene.meshes.begin());
		std::map<std::string, tinygltf::Mesh>::const_iterator itEnd(scene.meshes.end());

		// for each mesh
		for (; it != itEnd; it++) {
			const tinygltf::Mesh & mesh = it->second;

			//std::pair<std::map<std::string, std::vector<PrimitiveDevBufPointers>>::iterator, bool> res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
			auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
			std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

			// for each primitive
			for (size_t i = 0; i < mesh.primitives.size(); i++) {
				const tinygltf::Primitive &primitive = mesh.primitives[i];

				if (primitive.indices.empty())
					return;

				// TODO: ? now position, normal, etc data type is predefined
				VertexIndex* dev_indices;
				VertexAttributePosition* dev_position;
				VertexAttributeNormal* dev_normal;
				VertexAttributeTexcoord* dev_texcoord0;

				// ----------Indices-------------

				const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
				const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
				BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

				// !! assume type is SCALAR
				int n = 1;
				int numIndices = indexAccessor.count;
				int componentTypeByteSize = sizeof(VertexIndex);
				int byteLength = numIndices * n * componentTypeByteSize;

				dim3 numThreadsPerBlock(128);
				dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				hipMalloc(&dev_indices, byteLength);
				_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
					numIndices,
					(BufferByte*)dev_indices,
					dev_bufferView,
					indexAccessor.byteStride,
					indexAccessor.byteOffset,
					componentTypeByteSize);


				checkCUDAError("Set Index Buffer");

				//!!!!!!!!!TODO: delete test
				hipDeviceSynchronize();
				std::vector<VertexIndex> indicesWatch(numIndices);
				hipMemcpy(&indicesWatch.at(0), dev_indices, numIndices*sizeof(VertexIndex), hipMemcpyDeviceToHost);

				// ---------Primitive Info-------


				// !! LINE_STRIP is not supported in tinygltfloader
				int numPrimitives;
				PrimitiveType primitiveType;
				switch (primitive.mode) {
				case TINYGLTF_MODE_TRIANGLES:
					primitiveType = PrimitiveType::Triangle;
					numPrimitives = numIndices / 3;
					break;
				case TINYGLTF_MODE_TRIANGLE_STRIP:
					primitiveType = PrimitiveType::Triangle;
					numPrimitives = numIndices - 2;
					break;
				case TINYGLTF_MODE_TRIANGLE_FAN:
					primitiveType = PrimitiveType::Triangle;
					numPrimitives = numIndices - 2;
					break;
				case TINYGLTF_MODE_LINE:
					primitiveType = PrimitiveType::Line;
					numPrimitives = numIndices / 2;
					break;
				case TINYGLTF_MODE_LINE_LOOP:
					primitiveType = PrimitiveType::Line;
					numPrimitives = numIndices + 1;
					break;
				case TINYGLTF_MODE_POINTS:
					primitiveType = PrimitiveType::Point;
					numPrimitives = numIndices;
					break;
				default:
					// TODO: error
					break;
				};


				// ----------Attributes-------------

				//std::map<std::string, std::string>::const_iterator it(primitive.attributes.begin());
				auto it(primitive.attributes.begin());
				//std::map<std::string, std::string>::const_iterator itEnd(primitive.attributes.end());
				auto itEnd(primitive.attributes.end());

				int numVertices = 0;
				// for each attribute
				for (; it != itEnd; it++) {
					const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

					int n = 1;
					if (accessor.type == TINYGLTF_TYPE_SCALAR) {
						n = 1;
					}
					else if (accessor.type == TINYGLTF_TYPE_VEC2) {
						n = 2;
					}
					else if (accessor.type == TINYGLTF_TYPE_VEC3) {
						n = 3;
					}
					else if (accessor.type == TINYGLTF_TYPE_VEC4) {
						n = 4;
					}

					BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
					BufferByte ** dev_attribute = NULL;
					
					numVertices = accessor.count;
					int componentTypeByteSize;

					if (it->first.compare("POSITION") == 0) {
						componentTypeByteSize = sizeof(VertexAttributePosition);
						dev_attribute = (BufferByte**)&dev_position;
					} 
					else if (it->first.compare("NORMAL") == 0) {
						componentTypeByteSize = sizeof(VertexAttributeNormal);
						dev_attribute = (BufferByte**)&dev_normal;
					}
					else if (it->first.compare("TEXCOORD_0") == 0) {
						componentTypeByteSize = sizeof(VertexAttributeTexcoord);
						dev_attribute = (BufferByte**)&dev_texcoord0;
					}


					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					int byteLength = numVertices * n * componentTypeByteSize;
					hipMalloc(dev_attribute, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numVertices,
						*dev_attribute,
						dev_bufferView,
						accessor.byteStride,
						accessor.byteOffset,
						componentTypeByteSize);

					std::string msg = "Set Attribute Buffer: " + it->first;
					checkCUDAError(msg.c_str());
				}

				// malloc for VertexOut
				VertexOut* dev_vertexOut;
				hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
				checkCUDAError("Malloc VertexOut Buffer");

				// ----------Materials-------------
				// TODO


				// at the end of the for loop of primitive
				// push dev pointers to map
				primitiveVector.push_back(PrimitiveDevBufPointers{
					primitive.mode,
					primitiveType,
					numPrimitives,
					numIndices,
					numVertices,

					dev_indices,
					dev_position,
					dev_normal,
					dev_texcoord0,

					dev_vertexOut	//VertexOut
				});

				totalNumPrimitives += numPrimitives;

			} // for each primitive

		} // for each mesh

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	



	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



/**
* for one primitive
* ?? can combine with pritimitiveAssembly to make only one kernel call??
*/
__global__ 
void _vertexTransformAndAssembly(int numVertices, PrimitiveDevBufPointers primitive, glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, int width, int height) {
	// TODO: delete for assignments

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		primitive.dev_verticesOut[vid].pos = MVP * glm::vec4(primitive.dev_position[vid], 1.0f);
		glm::vec4 temp = MV * glm::vec4(primitive.dev_position[vid], 1.0f);
		primitive.dev_verticesOut[vid].eyePos = glm::vec3(temp) / temp.w;
		primitive.dev_verticesOut[vid].eyeNor = MV_normal * primitive.dev_normal[vid];
		//primitive.dev_verticesOut[vid].texcoord0 = primitive.dev_texcoord0[vid];

		// clipping space (to NDC -1,1) to viewport
		glm::vec4 & pos = primitive.dev_verticesOut[vid].pos;
		pos.x = 0.5f * (float)width * (pos.x / pos.w + 1.0f);
		pos.y = 0.5f * (float)height * (pos.y / pos.w + 1.0f);
		pos.z = 0.5f * (pos.z / pos.w + 1.0f);

		//perspective correct interpolation
		primitive.dev_verticesOut[vid].texcoord0 = primitive.dev_texcoord0[vid] / pos.w;
	}
}



static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {
	// TODO: delete for assignments

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {
		int pid;	//id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}
	}
	
	// TODO: other primitive types
}











// -----------------------------------------------------------
// TODO: delete for assignment

struct Edge
{
	VertexOut v[2];

	float x, z;
	float dx, dz;


	//
	//VertexOut cur_v;	//used for interpolate between a scan line
	float gap_y;
};
//e.v[0] is the one with smaller y value
//scan from v[0] to v[1]
__device__
void constructEdge(Edge & e, const VertexOut & v0, const VertexOut & v1)
{
	if (v0.pos.y <= v1.pos.y)
	{
		e.v[0] = v0;
		e.v[1] = v1;
	}
	else
	{
		e.v[0] = v1;
		e.v[1] = v0;
	}

	e.gap_y = 0.0f;

}

__device__
float initEdge(Edge & e, float y)
{
	e.gap_y = e.v[1].pos.y - e.v[0].pos.y;

	e.dx = (e.v[1].pos.x - e.v[0].pos.x) / e.gap_y;
	e.dz = (e.v[1].pos.z - e.v[0].pos.z) / e.gap_y;
	e.x = e.v[0].pos.x + (y - e.v[0].pos.y) * e.dx;
	e.z = e.v[0].pos.z + (y - e.v[0].pos.y) * e.dz;

	return (y - e.v[0].pos.y) / e.gap_y;
}

__device__
void updateEdge(Edge & e)
{
	e.x += e.dx;
	e.z += e.dz;
}



__device__
void drawOneScanLine(int width, const Edge & e1, const Edge & e2, int y, 
	float u1, float u2, Fragment * fragments, int * depth, const Primitive & tri)
{

	// Find the starting and ending x coordinates and
	// clamp them to be within the visible region
	int x_left = (int)(ceilf(e1.x) + EPSILON);
	int x_right = (int)(ceilf(e2.x) + EPSILON);


	float x_left_origin = e1.x;
	float x_right_origin = e2.x;

	if (x_left < 0)
	{
		x_left = 0;
	}

	if (x_right > width)
	{
		x_right = width;
	}

	// Discard scanline with no actual rasterization and also
	// ensure that the length is larger than zero
	if (x_left >= x_right) { return; }


	//TODO: get two interpolated segment end points
	//VertexOut cur_v_e1 = interpolateVertexOut(e1.v[0], e1.v[1], u1);
	//VertexOut cur_v_e2 = interpolateVertexOut(e2.v[0], e2.v[1], u2);


	//Initialize attributes
	float dz = (e2.z - e1.z) / (e2.x - e1.x);
	float z = e1.z + (x_left_origin - e1.x) * dz;

	//Interpolate
	//printf("%d,%d\n", x_left, x_right);
	//float gap_x = x_right_origin - x_left_origin;
	for (int x = x_left; x < x_right; ++x)
	{

		int idx = x + y * width;

		//VertexOut p = interpolateVertexOut(cur_v_e1, cur_v_e2, ((float)x-x_left_origin) / gap_x);


		//using barycentric
		glm::vec3 t[3] = { glm::vec3(tri.v[0].pos), glm::vec3(tri.v[1].pos), glm::vec3(tri.v[2].pos) };
		glm::vec3 u = calculateBarycentricCoordinate(t, glm::vec2(x, y));

		VertexOut p;
		//p.pos = u.x * tri.v[0].pos + u.y * tri.v[1].pos + u.z * tri.v[2].pos;
		//p.pos.w = u.x * tri.v[0].pos.w + u.y * tri.v[1].pos.w + u.z * tri.v[2].pos.w;
		p.pos = u.x * tri.v[0].pos + u.y * tri.v[1].pos + u.z * tri.v[2].pos;
		p.eyeNor = u.x * tri.v[0].eyeNor + u.y * tri.v[1].eyeNor + u.z * tri.v[2].eyeNor;
		//p.pos.w = u.x * tri.v[0].pos.w + u.y * tri.v[1].pos.w + u.z * tri.v[2].pos.w;

		int z_int = (int)(z * INT_MAX);

		int* address = &depth[idx];

		atomicMin(address, z_int);

		if (*address == z_int)
		{
			//fragments[idx].depth = z;
			//fragments[idx].color = glm::vec3(p.pos.z);

			//fragments[idx].color = glm::vec3(p.pos.z);

			fragments[idx].color = p.eyeNor;

			//fragments[idx].color = glm::vec3(1.0f, 1.0f, 1.0f);

			//fragments[idx].has_fragment = true;

		}



		z += dz;
	}
}


/**
* Rasterize the area between two edges as the left and right limit.
* e1 - longest y span
*/
__device__
void drawAllScanLines(int width, int height, Edge  e1, Edge  e2, 
	Fragment * fragments, int * depth, const Primitive &  tri)
{
	// Discard horizontal edge as there is nothing to rasterize
	if (e2.v[1].pos.y - e2.v[0].pos.y == 0.0f) { return; }

	// Find the starting and ending y positions and
	// clamp them to be within the visible region
	int y_bot = (int)(ceilf(e2.v[0].pos.y) + EPSILON);
	int y_top = (int)(ceilf(e2.v[1].pos.y) + EPSILON);



	float y_bot_origin = ceilf(e2.v[0].pos.y);
	float y_top_origin = floorf(e2.v[1].pos.y);

	if (y_bot < 0)
	{
		y_bot = 0;

	}

	if (y_top > height)
	{
		y_top = height;
	}


	//Initialize edge's structure
	float u1_base = initEdge(e1, y_bot_origin);
	initEdge(e2, y_bot_origin);


	//printf("%f,%f\n", e1.v[0].uv.x / e1.v[0].divide_w_clip, e1.v[0].uv.y / e1.v[0].divide_w_clip );

	for (int y = y_bot; y < y_top; ++y)
	{

		float u2 = ((float)y - y_bot_origin) / e2.gap_y;
		float u1 = u1_base + ((float)y - y_bot_origin) / e1.gap_y;
		if (e1.x <= e2.x)
		{
			drawOneScanLine(width, e1, e2, y, u1, u2, fragments, depth, tri);
		}
		else
		{
			drawOneScanLine(width, e2, e1, y, u2, u1, fragments, depth, tri);
		}

		//update edge
		updateEdge(e1);
		updateEdge(e2);
	}
}

/**
* Each thread handles one triangle
* rasterization
*/
__global__
void kernScanLineForOneTriangle(int num_tri, int width, int height
, Primitive * triangles, Fragment * depth_fragment, int * depth)
{
	int triangleId = blockDim.x * blockIdx.x + threadIdx.x;

	if (triangleId >= num_tri)
	{
		return;
	}


	Primitive tri = triangles[triangleId];	//copy


	


	bool outside = true;

	//currently tri.v are in clipped coordinates
	//need to transform to viewport coordinate
	for (int i = 0; i < 3; i++)
	{


		////////
		if (tri.v[i].pos.x < (float)width && tri.v[i].pos.x >= 0
			&& tri.v[i].pos.y < (float)height && tri.v[i].pos.y >= 0)
		{
			outside = false;
			// test------------------------------------
			int idx = tri.v[i].pos.x + tri.v[i].pos.y * width;
			depth_fragment[idx].color = glm::vec3(1.0f, 1.0f, 1.0f);
			//printf("%d", triangleId);
		}
	}


	//discard triangles that are totally out of the viewport
	if (outside)
	{
		return;
	}
	/////




	


	//build edge
	// for line scan
	Edge edges[3];

	constructEdge(edges[0], tri.v[0], tri.v[1]);
	constructEdge(edges[1], tri.v[1], tri.v[2]);
	constructEdge(edges[2], tri.v[2], tri.v[0]);


	//Find the edge with longest y span
	float maxLength = 0.0f;
	int longEdge = -1;
	for (int i = 0; i < 3; ++i)
	{
		float length = edges[i].v[1].pos.y - edges[i].v[0].pos.y;
		if (length > maxLength)
		{
			maxLength = length;
			longEdge = i;
		}
	}


	// get indices for other two shorter edges
	int shortEdge0 = (longEdge + 1) % 3;
	int shortEdge1 = (longEdge + 2) % 3;

	// Rasterize two parts separately
	drawAllScanLines(width, height, edges[longEdge], edges[shortEdge0], depth_fragment, depth, tri);
	drawAllScanLines(width, height, edges[longEdge], edges[shortEdge1], depth_fragment, depth, tri);



}










/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// TODO: Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");



		// test, copy back to host memory
		std::vector<Primitive> hst_primitives(totalNumPrimitives);
		hipMemcpy(&hst_primitives.at(0), dev_primitives, totalNumPrimitives * sizeof(Primitive), hipMemcpyDeviceToHost);
		checkCUDAError("mem test");

		
		
	}
	
	// !!!!!!!!!!!!!!!!Rasterize: temp test
	hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	{
		dim3 numThreadsPerBlock(64);
		dim3 numBlocks((totalNumPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
		kernScanLineForOneTriangle << <numBlocks, numThreadsPerBlock >> >(totalNumPrimitives, width, height, dev_primitives, dev_depthbuffer, dev_depth);
	}



    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_verticesOut);
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
